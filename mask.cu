#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <thread>
#include <vector>
#include <stdexcept>


#ifdef _WIN32
#include <windows.h>
#include <mmsystem.h>
#pragma comment(lib, "winmm.lib")  // Link with winmm.lib
#endif


#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"


#define SCREEN_WIDTH 2560
#define SCREEN_HEIGHT 1440 

#define DEBUG_MODE false
#define NO_SOUNDS true

struct Event {
    const char* name;
    float last_time;
    const char* message;
    const char* example_img_path;
    const char* sound_path;
    unsigned char* image;
    const int top_left[2];
    const int bottom_right[2];
    const unsigned char threshold;
    const int diff_threshold;
    const float delay;
};

unsigned char* loadImage(const char* filename, int* width, int* height) {
    int channels;
    unsigned char* img = stbi_load(filename, width, height, &channels, 3);
    if (!img) {
        printf("Error loading image %s\n", filename);
        return nullptr;
    }
    return img;
}

void saveImage(const char* filename, const unsigned char* image, int width, int height, bool is_grayscale=true) {

 if (!stbi_write_png(filename, 
                     width, 
                     height, 
                     is_grayscale ? 1 : 3,  // 1 channel (grayscale) or 3 channels (RGB)
                     image, 
                     width * (is_grayscale ? 1 : 3))) {  // stride = width for grayscale
     printf("Warning: Failed to save debug image: %s\n", filename);
 }
}

#ifdef _WIN32
unsigned char* captureScreen(int* width, int* height) {
    // Use the constants instead of GetSystemMetrics
    *width = SCREEN_WIDTH;   // 2560
    *height = SCREEN_HEIGHT; // 1440

    // Create device context and bitmap
    HDC hScreenDC = GetDC(NULL);
    if (!hScreenDC) {
        printf("Error: GetDC failed with error code %lu\n", GetLastError());
        return nullptr;
    }

    HDC hMemoryDC = CreateCompatibleDC(hScreenDC);
    if (!hMemoryDC) {
        printf("Error: CreateCompatibleDC failed with error code %lu\n", GetLastError());
        ReleaseDC(NULL, hScreenDC);
        return nullptr;
    }

    HBITMAP hBitmap = CreateCompatibleBitmap(hScreenDC, *width, *height);
    if (!hBitmap) {
        printf("Error: CreateCompatibleBitmap failed with error code %lu\n", GetLastError());
        DeleteDC(hMemoryDC);
        ReleaseDC(NULL, hScreenDC);
        return nullptr;
    }

    HBITMAP hOldBitmap = (HBITMAP)SelectObject(hMemoryDC, hBitmap);
    if (!hOldBitmap) {
        printf("Error: SelectObject failed with error code %lu\n", GetLastError());
        DeleteObject(hBitmap);
        DeleteDC(hMemoryDC);
        ReleaseDC(NULL, hScreenDC);
        return nullptr;
    }

    // Copy screen to bitmap
    if (!BitBlt(hMemoryDC, 0, 0, *width, *height, hScreenDC, 0, 0, SRCCOPY)) {
        printf("Error: BitBlt failed with error code %lu\n", GetLastError());
        SelectObject(hMemoryDC, hOldBitmap);
        DeleteObject(hBitmap);
        DeleteDC(hMemoryDC);
        ReleaseDC(NULL, hScreenDC);
        return nullptr;
    }

    // Get bitmap info
    BITMAPINFOHEADER bi;
    ZeroMemory(&bi, sizeof(BITMAPINFOHEADER));
    bi.biSize = sizeof(BITMAPINFOHEADER);
    bi.biWidth = *width;
    bi.biHeight = -*height;  // Negative height for top-down image
    bi.biPlanes = 1;
    bi.biBitCount = 24;
    bi.biCompression = BI_RGB;

    // Calculate stride (bytes per row, must be DWORD-aligned)
    int stride = (*width * 3 + 3) & ~3;

    // Allocate memory for pixel data
    unsigned char* pixels = (unsigned char*)malloc(stride * *height);
    if (!pixels) {
        printf("Error: Failed to allocate memory for pixels\n");
        SelectObject(hMemoryDC, hOldBitmap);
        DeleteObject(hBitmap);
        DeleteDC(hMemoryDC);
        ReleaseDC(NULL, hScreenDC);
        return nullptr;
    }

    // Get pixel data
    int scanlines = GetDIBits(hMemoryDC, hBitmap, 0, *height, pixels, (BITMAPINFO*)&bi, DIB_RGB_COLORS);
    if (scanlines != *height) {
        printf("Error: GetDIBits failed. Expected %d scanlines, got %d. Error code: %lu\n", 
               *height, scanlines, GetLastError());
        free(pixels);
        SelectObject(hMemoryDC, hOldBitmap);
        DeleteObject(hBitmap);
        DeleteDC(hMemoryDC);
        ReleaseDC(NULL, hScreenDC);
        return nullptr;
    }

    // Cleanup
    SelectObject(hMemoryDC, hOldBitmap);
    DeleteObject(hBitmap);
    DeleteDC(hMemoryDC);
    ReleaseDC(NULL, hScreenDC);

    return pixels;
}

void playSound(const char* sound_path) {
    if (!PlaySoundA(sound_path, NULL, SND_FILENAME | SND_ASYNC)) {
        printf("Error playing sound: %lu\n", GetLastError());
    }
}
#endif


float getCurrentTime() {
    auto now = std::chrono::high_resolution_clock::now();
    auto duration = now.time_since_epoch();
    return std::chrono::duration_cast<std::chrono::milliseconds>(duration).count() / 1000.0f;
}

bool initCUDA() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        printf("Error getting device count: %s\n", hipGetErrorString(error));
        return false;
    }
    
    if (deviceCount == 0) {
        printf("No CUDA devices found\n");
        return false;
    }
    
    hipDeviceProp_t deviceProp;
    error = hipGetDeviceProperties(&deviceProp, 0);
    if (error != hipSuccess) {
        printf("Error getting device properties: %s\n", hipGetErrorString(error));
        return false;
    }
    
    printf("Using CUDA device: %s\n", deviceProp.name);
    printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    
    error = hipSetDevice(0);
    if (error != hipSuccess) {
        printf("Error setting device: %s\n", hipGetErrorString(error));
        return false;
    }
    
    return true;
}

__global__ void grayscaleThresholdDifferenceKernel(
    const unsigned char* input,
    unsigned char* output,
    const int width,
    const int height,
    const unsigned char threshold,
    const unsigned char* example_img,
    int* d_difference
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_pixels = width * height;
    
    if (idx >= total_pixels) return;
    
    const int x = idx % width;
    const int y = idx / width;
    const int rgb_idx = (y * width + x) * 3;
    
    // RGB to grayscale conversion
    const unsigned char gray = 0.299f * input[rgb_idx] + 
                      0.587f * input[rgb_idx + 1] + 
                      0.114f * input[rgb_idx + 2];
    
    // Thresholding
    const unsigned char thresholded = (threshold > 0 && gray > threshold) ? 255 : 0;
    output[idx] = thresholded;
    //Calculate difference
    if(d_difference == nullptr || example_img == nullptr) return;

    const unsigned char diff = abs(thresholded - example_img[idx]);
    atomicAdd(d_difference, diff);
}

void cropImage(const unsigned char* input, unsigned char* output, const int full_width, const int* top_left, const int* bottom_right) {
    int crop_width = bottom_right[0] - top_left[0];
    int crop_height = bottom_right[1] - top_left[1];

    // Copy cropped region
    for (int y = 0; y < crop_height; y++) {
        for (int x = 0; x < crop_width; x++) {
            int src_idx = ((y + top_left[1]) * full_width + (x + top_left[0])) * 3;
            int dst_idx = (y * crop_width + x) * 3;
            
            // Copy RGB channels
            output[dst_idx] = input[src_idx];        // R
            output[dst_idx + 1] = input[src_idx + 1];// G 
            output[dst_idx + 2] = input[src_idx + 2];// B
        }
    }
}

void getCroppedGrayThreshImage(
    const unsigned char* input,
    unsigned char* output,
    const int full_width,
    const int* top_left,
    const int* bottom_right,
    const unsigned char threshold
) {
    int crop_width = bottom_right[0] - top_left[0];
    int crop_height = bottom_right[1] - top_left[1];
    int size = crop_width * crop_height;    
    
    unsigned char* processed_img = (unsigned char*)malloc(size * 3);  // Allocate for RGB cropped image
    cropImage(input, processed_img, full_width, top_left, bottom_right);

    // CUDA

    unsigned char* d_processed_img;
    hipMalloc(&d_processed_img, size*3);
    hipMemcpy(d_processed_img, processed_img, size*3, hipMemcpyHostToDevice);

    unsigned char* d_output;
    hipMalloc(&d_output, size);

    const int blockSize = 256;
    const int numBlocks = (size + blockSize - 1) / blockSize;
    grayscaleThresholdDifferenceKernel<<<numBlocks, blockSize>>>(
        d_processed_img,
        d_output,
        crop_width,
        crop_height,
        threshold,
        nullptr,
        nullptr
    );
   
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_processed_img);
    hipFree(d_output);

    free(processed_img);
}

bool isImgsMatch(
    const unsigned char* img,
    const unsigned char* example_img,
    const int full_width,
    const int* top_left,
    const int* bottom_right,
    const int diff_threshold,
    const unsigned char threshold,
    int* output_difference = nullptr
) {
    int crop_width = bottom_right[0] - top_left[0];
    int crop_height = bottom_right[1] - top_left[1];
    int size = crop_width * crop_height;    
    
    unsigned char* processed_img = (unsigned char*)malloc(size * 3);  // Allocate for RGB cropped image
    cropImage(img, processed_img, full_width, top_left, bottom_right);

    unsigned char* output = (unsigned char*)malloc(size);  // Allocate for grayscale cropped image
    int difference = 0;

    // CUDA

    int* d_difference;
    hipMalloc(&d_difference, sizeof(int));
    hipMemset(d_difference, 0, sizeof(int));

    unsigned char* d_processed_img;
    hipMalloc(&d_processed_img, size*3);
    hipMemcpy(d_processed_img, processed_img, size*3, hipMemcpyHostToDevice);

    unsigned char* d_example_img;
    hipMalloc(&d_example_img, size);
    hipMemcpy(d_example_img, example_img, size, hipMemcpyHostToDevice);

    unsigned char* d_output;
    hipMalloc(&d_output, size);

    const int blockSize = 256;
    const int numBlocks = (size + blockSize - 1) / blockSize;
    grayscaleThresholdDifferenceKernel<<<numBlocks, blockSize>>>(
        d_processed_img,
        d_output,
        crop_width,
        crop_height,
        threshold,
        d_example_img,
        d_difference
    );

    hipMemcpy(&difference, d_difference, sizeof(int), hipMemcpyDeviceToHost);    
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_processed_img);
    hipFree(d_example_img);
    hipFree(d_difference);
    hipFree(d_output);


    if(DEBUG_MODE){
        // Save debug images
        char debug_filename[256];
        snprintf(debug_filename, sizeof(debug_filename), "./images/debug/%s.png", "example_img");
        saveImage(debug_filename, example_img, crop_width, crop_height, true);
        saveImage("./images/debug/processed.png", output, crop_width, crop_height, true);
    }

    free(output);
    free(processed_img);
    
    difference = difference / 255;
    if (output_difference != nullptr) {
        *output_difference = difference;
    }
    if (DEBUG_MODE) printf("%d %d\n", difference, diff_threshold);
    return difference < diff_threshold;
}

std::vector<Event> initEvents() {
    // Initialize events
    std::vector<Event> events = {
        {
            "PLANT",
            0.0f,
            "!!!Bomb has been planted!!!",
            "./images/planted_example.png",
            "./sounds/planted.wav",
            nullptr,
            {1012, 1070},
            {1547, 1149},
            127,
            1000,
            4.0f
        },
        {
            "WON",
            0.0f,
            "!!!Win!!!",
            "./images/won_example.png",
            "./sounds/won.wav",
            nullptr,
            {880, 250},
            {1740, 360},
            127,
            3000,
            10.0f
        },
        {
            "1KILL",
            0.0f,
            "!!!1st kill!!!",
            "./images/kill1_example.png",
            "./sounds/kill1.wav",
            nullptr,
            {1150, 1200},
            {1325, 1332},
            180,
            2000,
            0.5f
        }
    };

    printf("Loading and processing example images...\n");
    // Load and process example images
    unsigned char* example_img = nullptr;
    int width, height;
    for (auto& event : events) {
        printf("Loading image: %s\n", event.example_img_path);
        example_img = loadImage(event.example_img_path, &width, &height);
        printf("Loaded image dimensions: %dx%d\n", width, height);

        // Process example image
        int crop_width = event.bottom_right[0] - event.top_left[0];
        int crop_height = event.bottom_right[1] - event.top_left[1];
        int size = crop_width * crop_height;
        event.image = (unsigned char*)malloc(size); // allocate croppedGrayThreshImage

        getCroppedGrayThreshImage(
            example_img,
            event.image,
            width,
            event.top_left,
            event.bottom_right,
            event.threshold
        );
    }
    free(example_img);
    return events;
}

bool isEventMatch(
    const unsigned char* img,
    const Event& event,
    int* output_difference = nullptr
){
    return isImgsMatch(
        img,
        event.image,
        SCREEN_WIDTH,
        event.top_left,
        event.bottom_right,
        event.diff_threshold,
        event.threshold,
        output_difference
    );
}

bool testEventWithImage(const std::vector<Event>& events, const char* event_name, const std::string& image_path) {
    // Test event matching on a single image
    const Event& event = *std::find_if(events.begin(), events.end(), 
        [event_name](const Event& e) { return strcmp(e.name, event_name) == 0; });

    int width, height;
    unsigned char* test_image = loadImage(image_path.c_str(), &width, &height);
        if (!test_image) {
        printf("Failed to load image: %s\n", image_path.c_str());
        free(test_image);
        throw std::runtime_error("Failed to load image");
    }

    bool matched = isEventMatch(test_image, event);
    free(test_image);
    
    printf("Matched: %s\n", matched ? "MATCHED" : "NOT MATCHED");

    return matched;
}

std::pair<std::vector<std::string>, std::vector<int>> testEventWithImages(const std::vector<Event>& events, const char* event_name, const std::string& test_dir_prefix) {
    // Test event matching on a directory of images
    const Event& event = *std::find_if(events.begin(), events.end(), 
        [event_name](const Event& e) { return strcmp(e.name, event_name) == 0; });

    std::string test_dir = test_dir_prefix + event_name + "/";
    std::vector<std::string> image_files;
    
    WIN32_FIND_DATA findData;
    HANDLE hFind = FindFirstFile((test_dir + "*.png").c_str(), &findData);
    if (hFind != INVALID_HANDLE_VALUE) {
        do {
            image_files.push_back(test_dir + findData.cFileName);
        } while (FindNextFile(hFind, &findData));
        FindClose(hFind);
    }

    printf("Found %zu images to test\n", image_files.size());

    std::vector<std::string> notMatchedPaths;
    std::vector<int> notMatchedDifferences;

    int img_width, img_height;
    unsigned char* test_image = nullptr;
    for (const auto& image_path : image_files) {
        test_image = loadImage(image_path.c_str(), &img_width, &img_height);
        
        if (!test_image) {
            printf("Failed to load image: %s\n", image_path.c_str());
            continue;
        }
        int* difference = (int*)malloc(sizeof(int));
        bool matched = isEventMatch(test_image, event, difference);
        if(!matched) {
            notMatchedPaths.push_back(image_path);
            notMatchedDifferences.push_back(*difference);
        }
        free(difference);
        free(test_image);
    }

    printf("Found %zu not matched images\n", notMatchedPaths.size());
    for (size_t i = 0; i < notMatchedPaths.size(); i++) {
        printf("%s (%d)\n", notMatchedPaths[i].c_str(), notMatchedDifferences[i]);
    }

    return std::make_pair(notMatchedPaths, notMatchedDifferences);
}

void pickNewGoodExamples(const char* event_name, const std::vector<std::string>& notMatchedPaths, const std::vector<int>& notMatchedDifferences) {
    std::vector<int> filteredDifferences;
    std::vector<std::string> filteredPaths;
    
    // Filter out very similar images
    for (size_t i = 0; i < notMatchedDifferences.size(); i++) {
        bool is_unique_with_margin = true;
        for (size_t j = 0; j < filteredDifferences.size(); j++) {
            if (abs(notMatchedDifferences[i] - filteredDifferences[j]) < 100) {
                is_unique_with_margin = false;
                break;
            }
        }
        if (is_unique_with_margin) {
            filteredDifferences.push_back(notMatchedDifferences[i]);
            filteredPaths.push_back(notMatchedPaths[i]);
        }
    }
    
    // Create examples directory if it doesn't exist
    std::string examples_dir = "./images/examples/" + std::string(event_name) + "/";
    CreateDirectory("./images", NULL);
    CreateDirectory("./images/examples", NULL);
    CreateDirectory(examples_dir.c_str(), NULL);

    printf("\nManual filtering of not matched images:\n");
    printf("Press ENTER to keep image, BACKSPACE to skip\n");


    char fullPath[MAX_PATH];
    for (size_t i = 0; i < filteredPaths.size(); i++) {
        const auto& path = filteredPaths[i];
        const std::string window_title = path.substr(path.find_last_of("/\\") + 1);
        
        // Get the full path
        if (GetFullPathNameA(path.c_str(), MAX_PATH, fullPath, nullptr) == 0) {
            printf("Error getting full path for: %s\n", path.c_str());
            continue;
        }
        
        // Open the image using the default image viewer
        HINSTANCE result = ShellExecuteA(
            NULL,           // No parent window
            "open",         // Operation
            fullPath,       // File path
            NULL,          // Parameters
            NULL,          // Working directory
            SW_SHOWMAXIMIZED  // Show window maximized (fullscreen)
        );
        
        printf("Reviewing: %s (%zu/%zu)\n", path.c_str(), i + 1, filteredPaths.size());
        
        bool validKey = false;
        while (!validKey) {
            if (GetAsyncKeyState(VK_RETURN) & 0x8000) {  // ENTER key
                // Copy file to examples directory
                std::string filename = path.substr(path.find_last_of("/\\") + 1);
                std::string dest_path = "./images/examples/" + std::string(event_name) + "/" + filename;
                if (CopyFileA(fullPath, dest_path.c_str(), FALSE)) {
                    printf("Copied to: %s\n", dest_path.c_str());
                } else {
                    printf("Failed to copy file. Error: %lu\n", GetLastError());
                }
                validKey = true;
                Sleep(200);
            }
            else if (GetAsyncKeyState(VK_BACK) & 0x8000) {  // BACKSPACE key
                printf("Skipped\n");
                validKey = true;
                Sleep(200);
            }
            Sleep(10);
        }

        // Close the image viewer window
        HWND hwnd = FindWindowA(NULL, window_title.c_str());
        if (hwnd != NULL) {
            PostMessage(hwnd, WM_CLOSE, 0, 0);
        }
    }

}

void detectEventsOnScreen(std::vector<Event>& events) {
    bool running = true;
    int errorCount = 0;
    const int MAX_ERRORS = 5;
    int width, height;
    unsigned char* screenshot = nullptr;

    printf("Starting main loop. Press ESC to exit.\n");
    while (running && errorCount < MAX_ERRORS) {
        try {
            float current_time = getCurrentTime();
            try{
                screenshot = captureScreen(&width, &height);
            }
            catch(const std::exception& e){
                free(screenshot);
                throw e;
            }

            for (auto& event : events) {
                bool is_match = isEventMatch(
                    screenshot,
                    event
                );
                
                if (is_match) {
                    float time_since_last = current_time - event.last_time;
                    if (time_since_last > event.delay) {
                        printf("%s\n", event.message);
                        if (!NO_SOUNDS && event.sound_path != nullptr) {
                           playSound(event.sound_path);
                        }
                        event.last_time = current_time;
                    }
                }
            }
          
            if (GetAsyncKeyState(VK_ESCAPE) & 0x8000) {
                printf("ESC pressed, exiting...\n");
                running = false;
            }
            
            std::this_thread::sleep_for(std::chrono::milliseconds(16));
        }
        catch (const std::exception& e) {
            printf("Main loop error: %s (%d/%d)\n", e.what(), ++errorCount, MAX_ERRORS);
            std::this_thread::sleep_for(std::chrono::milliseconds(1000));
        }
    }
    free(screenshot);
}

int main() {
    printf("Program starting...\n");
    printf("Initializing CUDA...\n");
    
    if (!initCUDA()) {
        printf("Failed to initialize CUDA. Press Enter to exit...\n");
        getchar();
            return 1;
        }
    
    printf("Screen resolution set to: %dx%d\n", SCREEN_WIDTH, SCREEN_HEIGHT);

    std::vector<Event> events = initEvents();

    //testEventWithImage(events, "PLANT", "./images/Counter-strike 2 2024.11.17 - 23.00.49.02/PLANT/frame_318.641.png");
    //std::pair<std::vector<std::string>, std::vector<int>> notMatched = testEventWithImages(events, "1KILL", "./images/Counter-strike 2 2024.11.17 - 23.00.49.02/");
    //pickNewGoodExamples("1KILL", notMatched.first, notMatched.second);
    detectEventsOnScreen(events);

    // Cleanup and exit
    for (auto& event : events) {
        free(event.image);
    }

    hipDeviceReset();
    return 0;
}
